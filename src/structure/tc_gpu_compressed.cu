// Copyright (c) 2022 MIT
// Author: Xuhao Chen
#include "graph.h"
void triangle_count(Graph &g, uint64_t &total);
template <bool use_segment> void triangle_count_cgr(Graph &g, uint64_t &total);
template <bool use_segment> void triangle_count_vbyte(Graph &g, uint64_t &total, std::string scheme);
void triangle_count_compressed(Graph &g, uint64_t &total, vidType num_cached = 0);

void printusage(std::string bin) {
  std::cout << "Try " << bin << " -s name-of-scheme(cgr) " 
            << "-i ../../inputs/mico/dag-streamvbyte "
            << "[-o (oriented)] [-p (permutated)] [-d (degree_threshold)]\n";
}

int main(int argc,char *argv[]) {
  std::string schemename = "cgr";
  std::string filename = "";
  bool permutated = false;
  bool oriented = false;
  bool segmented = false;
  vidType degree_threshold = 32;
  int c;
  while ((c = getopt(argc, argv, "s:i:opgd:h")) != -1) {
    switch (c) {
      case 's':
        schemename = optarg;
        break;
      case 'i':
        filename = optarg;
        break;
      case 'o':
        oriented = true;
        break;
      case 'p':
        permutated = true;
        break;
      case 'g':
        segmented = true;
        break;
      case 'd':
        degree_threshold = atoi(optarg);
        break;
      case 'h':
        printusage(argv[0]);
        return 0;
      default:
        abort();
    }
  }
  if (argc < 3) {
    std::cout << "# arguments (" << argc << ") incorrect\n";
    printusage(argv[0]);
    return -1;
  }
  if (!oriented) {
    std::cout << "Graph must be oriented\n";
    printusage(argv[0]);
    return -1;
  }
 
  Graph g;
  g.set_degree_threshold(degree_threshold);
  if (schemename == "decomp")
    g.load_graph(filename);
  else
    g.load_compressed_graph(filename, schemename, permutated);
  g.print_meta_data();
  //g.print_graph();

  uint64_t total = 0;
  if (schemename == "decomp") {
    triangle_count(g, total);
  } else if (schemename == "cgr") {
    if (segmented)
      triangle_count_cgr<true>(g, total);
    else
      triangle_count_cgr<false>(g, total);
  } else { // vbyte, unary, hybrid
    if (segmented)
      triangle_count_vbyte<true>(g, total, schemename);
    else
      triangle_count_vbyte<false>(g, total, schemename);
  }
  std::cout << "total_num_triangles = " << total << "\n";
  return 0;
}

#include <hipcub/hipcub.hpp>
typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;

#include "graph_gpu.h"
#include "cuda_launch_config.hpp"
#include "triangle_bs_warp_vertex.cuh"
void triangle_count(Graph &g, uint64_t &total) {
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  GraphGPU gg(g);
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (g.V()-1)/WARPS_PER_BLOCK+1;
  if (nblocks > 65536) nblocks = 65536;
  refine_kernel_config(nthreads, nblocks, triangle_bs_warp_vertex);
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  Timer t;
  t.Start();
  triangle_bs_warp_vertex<<<nblocks, nthreads>>>(0, g.V(), gg, d_total);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "runtime [tc_gpu_base] = " << t.Seconds() << " sec\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
}

#include "graph_gpu_compressed.h"
typedef GraphGPUCompressed GraphTy;
#include "triangle_bs_warp_vertex_cgr.cuh"
template <bool use_segment>
void triangle_count_cgr(Graph &g, uint64_t &total) {
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (g.V()-1)/WARPS_PER_BLOCK+1;
  if (nblocks > 65536) nblocks = 65536;
  refine_kernel_config(nthreads, nblocks, triangle_bs_warp_vertex_cgr<use_segment>);

  std::cout << "Allocating buffer for decompressed adjacency lists\n";
  vidType *buffer;
  size_t num_per_block = WARPS_PER_BLOCK;
  allocate_gpu_buffer(3 * size_t(g.get_max_degree()) * num_per_block * nblocks, buffer);

  std::cout << "Allocating the graph on GPU\n";
  GraphGPUCompressed gg(g, "cgr", g.get_degree_threshold());
  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  Timer t;
  t.Start();
  triangle_bs_warp_vertex_cgr<use_segment><<<nblocks, nthreads>>>(0, g.V(), gg, buffer, d_total);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "runtime [tc_gpu_cgr] = " << t.Seconds() << " sec\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
  CUDA_SAFE_CALL(hipFree(buffer));
}

#include "triangle_bs_warp_vertex_vbyte.cuh"
#include "triangle_bs_warp_vertex_unary.cuh"
#include "triangle_bs_warp_vertex_hybrid.cuh"
template <bool use_segment>
void triangle_count_vbyte(Graph &g, uint64_t &total, std::string scheme) {
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (g.V()-1)/WARPS_PER_BLOCK+1;
  if (nblocks > 65536) nblocks = 65536;
  if (scheme == "unary") {
    refine_kernel_config(nthreads, nblocks, triangle_bs_warp_vertex_unary);
  } else if (scheme == "hybrid") {
    refine_kernel_config(nthreads, nblocks, triangle_bs_warp_vertex_hybrid<use_segment>);
  } else if (scheme == "streamvbyte") {
    refine_kernel_config(nthreads, nblocks, triangle_bs_warp_vertex_vbyte<0,true>);
  } else {
    refine_kernel_config(nthreads, nblocks, triangle_bs_warp_vertex_vbyte<1,true,4>);
  }

  std::cout << "Allocating buffer for decompressed adjacency lists\n";
  vidType *buffer;
  size_t num_per_block = WARPS_PER_BLOCK;
  allocate_gpu_buffer(3 * size_t(g.get_max_degree()) * num_per_block * nblocks, buffer);

  std::cout << "Allocating the graph on GPU\n";
  GraphGPUCompressed gg(g, scheme, g.get_degree_threshold());
  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  Timer t;
  t.Start();
  if (scheme == "unary") {
    std::cout << "launching unary kernel\n";
    triangle_bs_warp_vertex_unary<<<nblocks, nthreads>>>(0, g.V(), gg, buffer, d_total);
  } else if (scheme == "hybrid") {
    std::cout << "launching hybrid kernel\n";
    triangle_bs_warp_vertex_hybrid<use_segment><<<nblocks, nthreads>>>(0, g.V(), gg, buffer, d_total);
  } else if (scheme == "streamvbyte") {
    std::cout << "launching streamvbyte kernel\n";
    triangle_bs_warp_vertex_vbyte<0,true><<<nblocks, nthreads>>>(0, g.V(), gg, buffer, d_total);
  } else {
    triangle_bs_warp_vertex_vbyte<1,true,4><<<nblocks, nthreads>>>(0, g.V(), gg, buffer, d_total);
  }
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "runtime [tc_gpu_" << scheme << "] = " << t.Seconds() << " sec\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
  CUDA_SAFE_CALL(hipFree(buffer));
}

#define VERTEX_PARALLEL
#define WARP_CENTRIC
#define USE_ZERO_COPY 1
#ifdef VERTEX_PARALLEL
#include "triangle_bs_warp_vertex_compressed.cuh"
#else
#include "triangle_bs_warp_edge_compressed.cuh"
#endif
void triangle_count_compressed(Graph &g, uint64_t &total, vidType num_cached) {
  size_t memsize = print_device_info(0);
#ifndef VERTEX_PARALLEL
  if (!USE_ZERO_COPY && g.is_compressed_only()) g.decompress();
#endif
  GraphGPUCompressed gg(g, "cgr");

  // kernel launch configuration
  size_t nthreads = BLOCK_SIZE, nblocks = (g.V()-1)/nthreads+1;
  if (nblocks > 65536) nblocks = 65536;
#ifdef VERTEX_PARALLEL
#ifdef USE_HINDEX
  refine_kernel_config(nthreads, nblocks, hindex_warp_vertex_compressed);
#else
  refine_kernel_config(nthreads, nblocks, bs_warp_vertex_compressed);
  //refine_kernel_config(nthreads, nblocks, bs_warp_vertex_compressed_cached);
#endif
#else
  auto nnz = gg.init_edgelist(g, 0, 0, USE_ZERO_COPY); // streaming edgelist using zero-copy
  nblocks = (nnz-1)/WARPS_PER_BLOCK+1;
  if (nblocks > 65536) nblocks = 65536;
#ifdef USE_HINDEX
  refine_kernel_config(nthreads, nblocks, hindex_warp_edge_compressed);
#else
  refine_kernel_config(nthreads, nblocks, bs_warp_edge_compressed);
#endif
#endif
  std::cout << "CUDA kernel (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  std::cout << "Allocating buffer for decompressed adjacency lists\n";
  vidType *buffer;
  size_t num_per_block = WARPS_PER_BLOCK;
#ifndef WARP_CENTRIC
  num_per_block = 1;
#endif
  allocate_gpu_buffer(3 * size_t(g.get_max_degree()) * num_per_block * nblocks, buffer);

#ifdef USE_HINDEX
  std::cout << "Allocating buckets for the hash map\n";
  vidType *bins;
  allocate_gpu_buffer(NUM_BUCKETS * BUCKET_SIZE * WARPS_PER_BLOCK * nblocks, bins);
  auto bins_mem = NUM_BUCKETS * BUCKET_SIZE * WARPS_PER_BLOCK * nblocks * sizeof(vidType);
  CUDA_SAFE_CALL(hipMemset(bins, 0, bins_mem));
#endif

  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  Timer t;
  t.Start();
#ifdef USE_HINDEX
  std::cout << "Using HINDEX ";
#ifdef VERTEX_PARALLEL
  std::cout << "vertex-parallel ";
#ifdef WARP_CENTRIC
  std::cout << "warp-centric\n";
  hindex_warp_vertex_compressed<<<nblocks, nthreads>>>(gg, bins, buffer, d_total);
#else // cta centric
  std::cout << "cta-centric\n";
  hindex_cta_vertex_compressed<<<nblocks, nthreads>>>(gg, bins, buffer, d_total, num_cached);
#endif
#else // edge parallel
  std::cout << "edge-parallel ";
#ifdef WARP_CENTRIC
  std::cout << "warp-centric\n";
  hindex_warp_edge_compressed<<<nblocks, nthreads>>>(gg, bins, buffer, d_total);
  //hindex_warp_edge_compressed_cache<<<nblocks, nthreads>>>(gg, bins, buffer, d_total, num_cached);
#else // cta centric
  std::cout << "cta-centric\n";
  hindex_cta_edge_compressed<<<nblocks, nthreads>>>(gg, bins, buffer, d_total);
#endif
#endif // end vertex/edge parallel
#else // USE_BinarySearch
  std::cout << "Using BinarySearch ";
#ifdef VERTEX_PARALLEL
  std::cout << "vertex-parallel ";
#ifdef WARP_CENTRIC
  std::cout << "warp-centric\n";
  bs_warp_vertex_compressed<<<nblocks, nthreads>>>(gg, buffer, d_total);
  //bs_warp_vertex_compressed_cached<<<nblocks, nthreads>>>(gg, buffer, d_total, num_cached);
#else // cta centric
  std::cout << "cta-centric\n";
  bs_cta_vertex_compressed<<<nblocks, nthreads>>>(gg, buffer, d_total);
#endif
#else // edge parallel
  std::cout << "edge-parallel ";
#ifdef WARP_CENTRIC
  std::cout << "warp-centric\n";
  bs_warp_edge_compressed<<<nblocks, nthreads>>>(gg, buffer, d_total, num_cached);
#else // cta centric
  std::cout << "cta-centric\n";
  bs_cta_edge_compressed<<<nblocks, nthreads>>>(gg, buffer, d_total);
#endif // end cta/warp centric
#endif // end vertex/edge parallel
#endif // end hindex/binarySearch
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime = " << t.Seconds() << " sec\n";
  std::cout << "throughput = " << double(g.E()) / t.Seconds() / 1e9 << " billion Traversed Edges Per Second (TEPS)\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
  CUDA_SAFE_CALL(hipFree(buffer));
#ifdef USE_HINDEX
  CUDA_SAFE_CALL(hipFree(bins));
#endif
}
